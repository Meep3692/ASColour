#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <fstream>
#include <string>
#include <array>
#include <vector>
#include <iterator>
#include <cmath>
#include "CIEDE2000.h"
#include "bitmap_image.hpp"
#include <windows.h>

#define printableChars 95
#define startingChar 32

/** A color in CIELAB colorspace */
	struct LAB
	{
		/** Lightness */
		float l;
		/** Color-opponent a dimension */
		float a;
		/** Color-opponent b dimension */
		float b;
	};
	/** Convenience definition for struct LAB */
	using LAB = struct LAB;

/** 
 * @brief Inverse sRGB gamma correction, transforms R' to R 
 */
#define INVGAMMACORRECTION(t)	\
	(((t) <= 0.0404482362771076) ? \
	((t)/12.92) : pow(((t) + 0.055)/1.055, 2.4))

/** @brief XYZ color of the D65 white point */
#define WHITEPOINT_X	0.950456
#define WHITEPOINT_Y	1.0
#define WHITEPOINT_Z	1.088754

/** 
 * @brief CIE L*a*b* f function (used to convert XYZ to L*a*b*)
 * http://en.wikipedia.org/wiki/Lab_color_space
 */
#define LABF(t)	\
	((t >= 8.85645167903563082e-3) ? \
	pow(t,0.333333333333333) : (841.0/108.0)*(t) + (4.0/29.0))

//Convert rgb to lab
LAB rgb2lab(float R, float G, float B){
	
	//rgb 2 xyz
	R /= 255.0;
	G /= 255.0;
	B /= 255.0;
	
	R = INVGAMMACORRECTION(R);
	G = INVGAMMACORRECTION(G);
	B = INVGAMMACORRECTION(B);
	float X = (0.4123955889674142161*R + 0.3575834307637148171*G + 0.1804926473817015735*B);
	float Y = (0.2125862307855955516*R + 0.7151703037034108499*G + 0.07220049864333622685*B);
	float Z = (0.01929721549174694484*R + 0.1191838645808485318*G + 0.9504971251315797660*B);
	
	//XYZ 2 Lab
	X /= WHITEPOINT_X;
	Y /= WHITEPOINT_Y;
	Z /= WHITEPOINT_Z;
	X = LABF(X);
	Y = LABF(Y);
	Z = LABF(Z);
	float L = 116*Y - 16;
	float a = 500*(X - Y);
	float b = 200*(Y - Z);
	
	return {L, a, b};
}

//Represents a printable character and its colour
struct printable{
	int character;
	int consoleColour;
	LAB colour;
};
using Printable = struct printable;

//Convert a pixel
__global__
void convert(Printable *output, Printable* printables, LAB* image, int offset, int total){
	int index = blockDim.x * blockIdx.x + threadIdx.x + offset;//Get pixel index to process
	
	if(index > total)
		return;
	
	LAB lab1 = image[index];//Get colour value
	float minDiff = 500;//Set high min diff
	int minDiffIndex = 0;//Index of closest character
	for(int i = 0; i < (printableChars * 256); i++){//For everything printable to console
		
		LAB lab2 = printables[i].colour;//Colour of current char/colour combo
		
		//Big fat CIEDE2000 comparison (super slow gpu killer)
		
		// 
		// "For these and all other numerical/graphical 􏰀delta E00 values
		// reported in this article, we set the parametric weighting factors
		// to unity(i.e., k_L = k_C = k_H = 1.0)." (Page 27).
		 
		const float k_L = 1.0, k_C = 1.0, k_H = 1.0;
		const float deg360InRad = 6.283185307179586476925286766559;
		const float deg180InRad = 3.1415926535897932384626433832795;
		const float pow25To7 = 6103515625.0; // pow(25, 7) 
		
		//
		// Step 1 
		 
		// Equation 2 
		float C1 = sqrt((lab1.a * lab1.a) + (lab1.b * lab1.b));
		float C2 = sqrt((lab2.a * lab2.a) + (lab2.b * lab2.b));
		// Equation 3 
		float barC = (C1 + C2) / 2.0f;
		// Equation 4 
		float powbarc7 = barC * barC * barC * barC * barC * barC * barC;
		float G = 0.5f * (1 - sqrt(powbarc7 / (powbarc7 + pow25To7)));
		// Equation 5 
		float a1Prime = (1.0f + G) * lab1.a;
		float a2Prime = (1.0f + G) * lab2.a;
		// Equation 6 
		float CPrime1 = sqrt((a1Prime * a1Prime) + (lab1.b * lab1.b));
		float CPrime2 = sqrt((a2Prime * a2Prime) + (lab2.b * lab2.b));
		// Equation 7 
		float hPrime1;
		if (lab1.b == 0 && a1Prime == 0)
			hPrime1 = 0.0f;
		else {
			hPrime1 = atan2(lab1.b, a1Prime);
			// 
			// This must be converted to a hue angle in degrees between 0 
			// and 360 by addition of 2􏰏 to negative hue angles.
			 
			if (hPrime1 < 0)
				hPrime1 += deg360InRad;
		}
		float hPrime2;
		if (lab2.b == 0 && a2Prime == 0)
			hPrime2 = 0.0f;
		else {
			hPrime2 = atan2(lab2.b, a2Prime);
			// 
			// This must be converted to a hue angle in degrees between 0 
			// and 360 by addition of 2􏰏 to negative hue angles.
			 
			if (hPrime2 < 0)
				hPrime2 += deg360InRad;
		}
		
		//
		// Step 2
		 
		// Equation 8 
		float deltaLPrime = lab2.l - lab1.l;
		// Equation 9 
		float deltaCPrime = CPrime2 - CPrime1;
		// Equation 10 
		float deltahPrime;
		float CPrimeProduct = CPrime1 * CPrime2;
		if (CPrimeProduct == 0)
			deltahPrime = 0;
		else {
			// Avoid the fabs() call 
			deltahPrime = hPrime2 - hPrime1;
			if (deltahPrime < -deg180InRad)
				deltahPrime += deg360InRad;
			else if (deltahPrime > deg180InRad)
				deltahPrime -= deg360InRad;
		}
		// Equation 11 
		float deltaHPrime = 2.0f * sqrt(CPrimeProduct) *
			sin(deltahPrime / 2.0f);
		
		//
		// Step 3
		 
		// Equation 12 
		float barLPrime = (lab1.l + lab2.l) / 2.0f;
		// Equation 13 
		float barCPrime = (CPrime1 + CPrime2) / 2.0f;
		// Equation 14 
		float barhPrime, hPrimeSum = hPrime1 + hPrime2;
		if (CPrime1 * CPrime2 == 0) {
			barhPrime = hPrimeSum;
		} else {
			if (fabs(hPrime1 - hPrime2) <= deg180InRad)
				barhPrime = hPrimeSum / 2.0f;
			else {
				if (hPrimeSum < deg360InRad)
					barhPrime = (hPrimeSum + deg360InRad) / 2.0f;
				else
					barhPrime = (hPrimeSum - deg360InRad) / 2.0f;
			}
		}
		// Equation 15 
		float T = 1.0 - (0.17f * cos(barhPrime - 0.5235987756f)) +
			(0.24f * cos(2.0f * barhPrime)) +
			(0.32f * cos((3.0f * barhPrime) + 0.10471975512f )) - 
			(0.20f * cos((4.0f * barhPrime) - 1.0995574288f));
		// Equation 16 
		float deltaTheta = 0.5235987756f *
			exp(-(((barhPrime - 4.799655443f) / 0.436332313f) * ((barhPrime - 4.799655443f) / 0.436332313f)));
		// Equation 17 
		float temp17_1 = (barCPrime * barCPrime * barCPrime * barCPrime * barCPrime * barCPrime * barCPrime);
		float R_C = 2.0f * sqrt(temp17_1 /
			(temp17_1 + pow25To7));
		// Equation 18 
		float temp18_1 = ((barLPrime - 50.0f) * (barLPrime - 50));
		float S_L = 1 + ((0.015f * temp18_1) /
			sqrt(20 + temp18_1));
		// Equation 19 
		float S_C = 1 + (0.045f * barCPrime);
		// Equation 20 
		float S_H = 1 + (0.015f * barCPrime * T);
		// Equation 21 
		float R_T = (-sin(2.0f * deltaTheta)) * R_C;
		
		float asdf1 = (deltaLPrime / (k_L * S_L));
		float asdf2 = (deltaCPrime / (k_C * S_C));
		float asdf3 = (deltaHPrime / (k_H * S_H));
		// Equation 22
		float deltaE = sqrt(
			(asdf1 * asdf1) +
			(asdf2 * asdf2) +
			(asdf3 * asdf3) + 
			(R_T * (deltaCPrime / (k_C * S_C)) * (deltaHPrime / (k_H * S_H))));
		
		if(deltaE < minDiff){
			minDiff = deltaE;
			minDiffIndex = i;
		}
	}
	output[index] = printables[minDiffIndex];
}

int main(int argc, char *argv[])
{
	//Load charset image
	bitmap_image charsetImg("charset.bmp");
	//Initialize array for every printable character in every colour in VRAM
	Printable *printables;
	hipMallocManaged(&printables, printableChars * 256 * sizeof(Printable));
	
	//Handle for console
	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	
	for(int col = 0; col < 256; col++){//For every colour
		for(int c = 0; c < printableChars; c++){//And every character
			int rTotal = 0;
			int gTotal = 0;
			int bTotal = 0;
			for(int x = 0; x < 8; x++){//For every pixel in the character
				for(int y = 0; y < 8; y++){
					rgb_t colour;
					charsetImg.get_pixel((c + startingChar) * 8 + x, col * 8 + y, colour);//Get colour of pixel at that location on the character
					rTotal += colour.red;//Add to totals
					gTotal += colour.green;
					bTotal += colour.blue;
					
				}
			}
			/*SetConsoleTextAttribute(hConsole, (char)col);
			std::cout << chars[c];
			SetConsoleTextAttribute(hConsole, 15);
			*/
			LAB avgColour = rgb2lab((double)(rTotal / 64), (double)(gTotal / 64), (double)(bTotal / 64));//Convert average rgb into lab
			//std::cout << ": " << avgColour.l << " " << avgColour.a << " " << avgColour.b << std::endl;
			printables[c + printableChars * col] = {c + startingChar, col, avgColour};//Set character in printables
		}
	}
	std::cout << "Characters coloured" << std::endl;
	
	bitmap_image testImg(argv[1]);//Load image
	
	int width = testImg.width();//Get dimensions
	int height = testImg.height();
	
	int pixels = width * height;//Number of pixels in image
	
	//Allocate result array
	Printable *result;
	hipMallocManaged(&result, pixels * sizeof(Printable));
	
	//Array for pixel colour values in vram
	LAB* image;
	hipMallocManaged(&image, pixels * sizeof(LAB));
	
	//For every pixel
	for(int x = 0; x < width; x++){
		for(int y = 0; y < height; y++){
			rgb_t colour;
			testImg.get_pixel(x, y, colour);//Get the colour
			image[(y * width) + x] = rgb2lab(colour.red, colour.green, colour.blue);//Convert it to LAB and store in array
		}
	}
	int pixelsLeft = pixels;//Number of pixels yet to process
	while(pixelsLeft > 0){//While there are pixels to process
		int pixelsToDo;//Pixels to process in this round
		if(pixelsLeft < 4096)//Process 5000 unless we are near the end, then process the rest
			pixelsToDo = pixelsLeft;
		else
			pixelsToDo = 4096;
		
		std::cout << "Running " << pixelsToDo << " pixels from " << pixels - pixelsLeft << std::endl;
		
		int blockSize = 64;//1024 threads per block
		int numBlocks = (pixelsToDo + blockSize - 1) / blockSize;//However many blocks we need
		
		// Run kernel
		convert<<<numBlocks, blockSize>>>(result, printables, image, pixels - pixelsLeft, pixels);
		
		hipDeviceSynchronize();//Wait for kernels to finish
		pixelsLeft -= pixelsToDo;//Less pixels left
	}
	
	//bitmap_image outputImage(width * 8, height * 8);
	FILE* bm = fopen(argv[2], "wb");
	fputc('B', bm);
	fputc('M', bm);
	int size = 54 + (((int)((192.0f * width + 31.0f) / 32)) * 4) * height * 8;
	int padding = (((int)((192.0f * width + 31.0f) / 32)) * 4) - (width * 24);
	fputc(size & 0xff, bm);
	fputc((size & 0xff00) >> 8, bm);
	fputc((size & 0xff0000) >> 16, bm);
	fputc((size & 0xff000000) >> 24, bm);
	fwrite("\0\0\0\006\0\0\0", 1, 8, bm);
	fwrite("(\0\0\0", 1, 4, bm);
	fputc(width * 8 & 0xff, bm);
	fputc((width * 8 & 0xff00) >> 8, bm);
	fputc((width * 8 & 0xff0000) >> 16, bm);
	fputc((width * 8 & 0xff000000) >> 24, bm);
	fputc(height * 8 & 0xff, bm);
	fputc((height * 8 & 0xff00) >> 8, bm);
	fputc((height * 8 & 0xff0000) >> 16, bm);
	fputc((height * 8 & 0xff000000) >> 24, bm);
	fputc(1, bm);
	fputc(0, bm);
	fputc(24, bm);
	fputc(0, bm);
	fwrite("\0\0\0\0\0\0\0\00d\0\0\00d\0\0\0\0\0\0\0\0\0\0\0", 1, 24, bm);
	FILE* sauce;
	if(argc > 3){
		sauce = fopen(argv[3], "wb");
	}
	for(int y = height - 1; y >= 0; y--){//For every pixel
		//Bitmap
		for(int subY = 0; subY < 8; subY++){
			for(int x = 0; x < width; x++){
				int colour = result[x + (y * width)].consoleColour;//Get colour
				int character = result[x + (y * width)].character;//Get character
				for(int subX = 0; subX < 8; subX++){
					rgb_t color;
					charsetImg.get_pixel(character * 8 + subX, colour * 8 + 7 - subY, color);
					fputc(color.blue, bm);
					fputc(color.green, bm);
					fputc(color.red, bm);
				}
			}
			for(int i = 0; i < padding; i++){
				fputc('\0', bm);
			}
		}
	}
	fclose(bm);
	for(int y = 0; y < height; y++){//For every pixel
		for(int x = 0; x < width; x++){
			int colour = result[x + (y * width)].consoleColour;//Get colour
			int character = result[x + (y * width)].character;//Get character
			
			if(character == 0 || character == 7 || character == 8 || character == 9 || character == 10 || character == 13){
				character = 0x20;
			}
			
			SetConsoleTextAttribute(hConsole, colour);//Set colour
			std::cout << (char)character;//Print character
			
			//Bitmap output
			//bitmap_image characterImage(8, 8);
			//charsetImg.region((character - startingChar) * 8, colour * 8, 8, 8, characterImage);
			//outputImage.copy_from(characterImage, x * 8, y * 8);
			
			
			//Special sauce output
			if(argc > 3){
				fputc(1, sauce);
				fputc(colour, sauce);
				fputc(character, sauce);
			}
		}
		SetConsoleTextAttribute(hConsole, 15);//Set colour back to default(line break changing bg colour is really slow)
		std::cout << std::endl;//New line
		if(argc > 3){
			fputc(1, sauce);
			fputc(15, sauce);
			fputc(10, sauce);
		}
	}
	//outputImage.save_image(argv[2]);
	
	if(argc > 3)
		fclose(sauce);

	return 0;
}